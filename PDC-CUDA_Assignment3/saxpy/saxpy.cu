#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"


// return GB/sec
float GBPerSec(int bytes, float sec) {
  return static_cast<float>(bytes) / (1024. * 1024. * 1024.) / sec;
}

// CUDA kernel function to perform SAXPY operation
__global__ void saxpy_kernel(int N, float alpha, float* x, float* y, float* result) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
        result[index] = alpha * x[index] + y[index];
}

void saxpyCuda(int N, float alpha, float* xarray, float* yarray, float* resultarray) {
    int totalBytes = sizeof(float) * 3 * N;

    // Number of threads per block and number of blocks
    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Pointers for device memory
    float *device_x = nullptr, *device_y = nullptr, *device_result = nullptr;

    // Allocate device memory
    hipMalloc(&device_x, N * sizeof(float));
    hipMalloc(&device_y, N * sizeof(float));
    hipMalloc(&device_result, N * sizeof(float));

    // Copy input arrays from host to device
    hipMemcpy(device_x, xarray, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_y, yarray, N * sizeof(float), hipMemcpyHostToDevice);

    // Start timing
    double startTime = CycleTimer::currentSeconds();

    // Launch CUDA kernel
    saxpy_kernel<<<blocks, threadsPerBlock>>>(N, alpha, device_x, device_y, device_result);

    // Check for any CUDA errors
    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(errCode));
    }

    // Copy result from device to host
    hipMemcpy(resultarray, device_result, N * sizeof(float), hipMemcpyDeviceToHost);

    // End timing
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;
    printf("Effective BW by CUDA saxpy: %.3f ms\t[%.3f GB/s]\n", 1000.f * overallDuration, GBPerSec(totalBytes, overallDuration));

    // Free device memory
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_result);
}

void printCudaInfo() {

    // print out stats about the GPU in the machine.  Useful if
    // students want to know what GPU they are running on.

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
